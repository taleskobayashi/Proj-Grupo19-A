#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <time.h>

#define CHECK(r) {_check((r), __LINE__);}
using namespace cv;
using namespace std;

//Função que verifica caso houver erro na sincronização do kernel, qual é o erro
inline __host__ void PostKernelCall(void)
{
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if(errSync != hipSuccess)
		printf("\nSync kernel error: %s.", hipGetErrorString(errSync));
	if(errAsync != hipSuccess)
		printf("\nAsync kernel error: %s.\n", hipGetErrorString(errAsync));
}

int prime = 5;

Mat writeImage(Mat img, uchar *red, uchar *green, uchar *blue);

//função que checa se uma chamada de uma função de cuda teve falha e mostra qual a  falha
inline void _check(hipError_t r, int line)
{
  if (r != hipSuccess)
  {
    printf("CUDA error on line %d: %s\n", line, hipGetErrorString(r));
    exit(0);
  }
}

void getChannels(Mat image, uchar *red, uchar *green, uchar *blue)
{
    
    for(int i = 0; i < image.cols; i++)
    {
        for(int j = 0; j < image.rows; j++)
        {
            int index = i + j*image.cols;
            red[index] = image.at<Vec3b>(j,i)[2];
            green[index] = image.at<Vec3b>(j,i)[1];
            blue[index] = image.at<Vec3b>(j,i)[0];
        }
    }
}

//função que verifica se o computador testado possui um device compativel com CUDA
__host__ void deviceQuery()
{
	int n;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&n);
	if(n < 1){ cout << "Error no cuda devices found!" << endl;  exit(-1);  }

	hipGetDeviceProperties(&prop, 0);
	cout << prop.name << endl;
	cout << "Max threads per block allowed: " << prop.maxThreadsPerBlock << endl;
	cout << "Cuda support for: " << prop.major << "." << prop.minor << endl;
}

Mat writeImage(Mat img, uchar *red, uchar *green, uchar *blue)
{

    for(int i = 0; i < img.cols; i++)
    {
        for(int j = 0; j < img.rows; j++)
        {
            int index = i + j*img.cols;
            img.at<Vec3b>(j,i)[2] = red[index];
            img.at<Vec3b>(j,i)[1] = green[index];
            img.at<Vec3b>(j,i)[0] = blue[index];
        }
    }
    return img;
}

//função principal em CUDA
__global__ void ComputeKernel(uchar *red, uchar *green, uchar *blue,
					 uchar *out_red, uchar *out_green, uchar *out_blue, int width, int height, int pr)

{
	int size = width * height;
	int index = threadIdx.x + blockIdx.x*blockDim.x;//y +x*tamanholinha
	if(index < size) //processar apenas as threads com pixel correspondente, evitando threads fantasmas
	{
		//computa
		
		int listI[25];
		int listJ[25];
		int pixelI = index % width;
		int pixelJ = index / width;

		int amount = pr;
    	int far = (amount - 1)/2;
   		int r_i = pixelI - far;
    	int r_j, c = 0;
    	for(int i = 0; i < amount; i++)
    	{
        	r_j = pixelJ - far;
        	for(int j = 0; j < amount; j++)
        	{
            	if(r_i >= 0 && r_i < width && r_j >= 0 && r_j < height)
	            {
	                listI[c] = r_i;
	                listJ[c] = r_j;
	            }
	            else
	            {
	                listI[c] = -1;
	                listJ[c] = -1;
	            }
	            c++;
	            r_j++;
	        }
	        r_i ++;
	    }

	    int sum_r = 0, sum_g = 0, sum_b = 0;
	    int count = 0;
	    for(int i = 0; i < amount*amount; i++)
	    {
	        if(listI[i] == -1) continue;
	        int id = listI[i] + listJ[i] * width;
	        sum_r += red[id];
	        sum_g += green[id];
	        sum_b += blue[id];
	        count ++;
	    }

	    out_red[index] = sum_r/count;
	    out_green[index] = sum_g/count;
	    out_blue[index] = sum_b/count;					 
	}
}

int main(int argc, char **argv)
{
	clock_t tempoInicial, tempoFinal;
    double tempoGasto;
    tempoInicial = clock();
	deviceQuery();

	Mat image;
	image = imread("imagem.jpg", 1);
	uchar *h_red, *h_green, *h_blue, *d_red, *d_green, *d_blue;
    uchar *d_out_red, *d_out_green, *d_out_blue;



    int w = image.cols;
	int h = image.rows;
	int size = w*h;

	h_red = (uchar *)malloc(sizeof(uchar)*size);
	h_green = (uchar *)malloc(sizeof(uchar)*size);
	h_blue = (uchar *)malloc(sizeof(uchar)*size);

	CHECK(hipMalloc((void **)&d_red, sizeof(uchar)*size));
	CHECK(hipMalloc((void **)&d_green, sizeof(uchar)*size));
	CHECK(hipMalloc((void **)&d_blue, sizeof(uchar)*size));
	CHECK(hipMalloc((void **)&d_out_red, sizeof(uchar)*size));
	CHECK(hipMalloc((void **)&d_out_green, sizeof(uchar)*size));
	CHECK(hipMalloc((void **)&d_out_blue, sizeof(uchar)*size));

	getChannels(image, h_red, h_green, h_blue);

	CHECK(hipMemcpy(d_red, h_red, sizeof(uchar)*size, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_green, h_green, sizeof(uchar)*size, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_blue, h_blue, sizeof(uchar)*size, hipMemcpyHostToDevice));

	int t = size;

	//           <<<numBlocks,numThreads per block>>>
	ComputeKernel<<<(t + 127)/128, 128>>>(d_red, d_green, d_blue, d_out_red,
										 d_out_green, d_out_blue, w, h, prime);

	PostKernelCall();
	
	CHECK(hipMemcpy(h_red, d_out_red, sizeof(uchar)*size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(h_green, d_out_green, sizeof(uchar)*size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(h_blue, d_out_blue, sizeof(uchar)*size, hipMemcpyDeviceToHost));

    Mat newimg = writeImage(image.clone(), h_red, h_green, h_blue);

    hipFree(d_out_red);
    hipFree(d_out_green);
    hipFree(d_out_blue);
    hipFree(d_red);
    hipFree(d_green);
    hipFree(d_blue);

    tempoFinal = clock();
    tempoGasto = tempoFinal-tempoInicial;
    printf("Tempo em segundos: %.2f\n", tempoGasto/CLOCKS_PER_SEC);
    
	imshow("nome", newimg);
	waitKey(0);
   	

	return 0;
}
